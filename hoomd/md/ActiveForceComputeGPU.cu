#include "hip/hip_runtime.h"
// Copyright (c) 2009-2021 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

// Maintainer: joaander

#include "ActiveForceComputeGPU.cuh"
#include "EvaluatorConstraintEllipsoid.h"
#include "hoomd/RNGIdentifiers.h"
#include "hoomd/RandomNumbers.h"
#include "hoomd/TextureTools.h"
using namespace hoomd;

#include <assert.h>

/*! \file ActiveForceComputeGPU.cu
    \brief Declares GPU kernel code for calculating active forces forces on the GPU. Used by
   ActiveForceComputeGPU.
*/

//! Kernel for setting active force vectors on the GPU
/*! \param group_size number of particles
    \param d_index_array stores list to convert group index to global tag
    \param d_force particle force on device
    \param d_torque particle torque on device
    \param d_orientation particle orientation on device
    \param d_f_act particle active force unit vector
    \param d_t_act particle active torque unit vector
    \param P position of the ellipsoid constraint
    \param rx radius of the ellipsoid in x direction
    \param ry radius of the ellipsoid in y direction
    \param rz radius of the ellipsoid in z direction
    \param orientationLink check if particle orientation is linked to active force vector
*/
__global__ void gpu_compute_active_force_set_forces_kernel(const unsigned int group_size,
                                                           unsigned int* d_index_array,
                                                           Scalar4* d_force,
                                                           Scalar4* d_torque,
                                                           const Scalar4* d_pos,
                                                           const Scalar4* d_orientation,
                                                           const Scalar4* d_f_act,
                                                           const Scalar4* d_t_act,
                                                           const Scalar3 P,
                                                           const Scalar rx,
                                                           const Scalar ry,
                                                           const Scalar rz,
                                                           const unsigned int N)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= group_size)
        return;

    unsigned int idx = d_index_array[group_idx];
    Scalar4 posidx = __ldg(d_pos + idx);
    unsigned int type = __scalar_as_int(posidx.w);

    Scalar4 fact = __ldg(d_f_act + type);

    vec3<Scalar> f(fact.w * fact.x, fact.w * fact.y, fact.w * fact.z);
    quat<Scalar> quati(__ldg(d_orientation + idx));
    vec3<Scalar> fi = rotate(quati, f);
    d_force[idx] = vec_to_scalar4(fi, 0);

    Scalar4 tact = __ldg(d_t_act + type);

    vec3<Scalar> t(tact.w * tact.x, tact.w * tact.y, tact.w * tact.z);
    vec3<Scalar> ti = rotate(quati, t);
    d_torque[idx] = vec_to_scalar4(ti, 0);
    }

//! Kernel for adjusting active force vectors to align parallel to an ellipsoid surface constraint
//! on the GPU
/*! \param group_size number of particles
    \param d_index_array stores list to convert group index to global tag
    \param d_pos particle positions on device
    \param d_f_act particle active force unit vector
    \param P position of the ellipsoid constraint
    \param rx radius of the ellipsoid in x direction
    \param ry radius of the ellipsoid in y direction
    \param rz radius of the ellipsoid in z direction
*/
__global__ void gpu_compute_active_force_set_constraints_kernel(const unsigned int group_size,
                                                                unsigned int* d_index_array,
                                                                const Scalar4* d_pos,
                                                                Scalar4* d_orientation,
                                                                const Scalar4* d_f_act,
                                                                const Scalar3 P,
                                                                const Scalar rx,
                                                                const Scalar ry,
                                                                const Scalar rz)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= group_size)
        return;

    unsigned int idx = d_index_array[group_idx];
    Scalar4 posidx = __ldg(d_pos + idx);
    unsigned int type = __scalar_as_int(posidx.w);

    EvaluatorConstraintEllipsoid Ellipsoid(P, rx, ry, rz);
    Scalar3 current_pos = make_scalar3(posidx.x, posidx.y, posidx.z);

    Scalar3 norm_scalar3 = Ellipsoid.evalNormal(
        current_pos); // the normal vector to which the particles are confined.
    vec3<Scalar> norm = vec3<Scalar>(norm_scalar3);

    Scalar4 fact = __ldg(d_f_act + type);

    vec3<Scalar> f(fact.x, fact.y, fact.z);
    quat<Scalar> quati(__ldg(d_orientation + idx));
    vec3<Scalar> fi = rotate(quati, f);

    Scalar dot_prod = fi.x * norm.x + fi.y * norm.y + fi.z * norm.z;

    Scalar dot_perp_prod = slow::sqrt(1 - dot_prod * dot_prod);

    Scalar phi_half = slow::atan(dot_prod / dot_perp_prod) / 2.0;

    fi.x -= norm.x * dot_prod;
    fi.y -= norm.y * dot_prod;
    fi.z -= norm.z * dot_prod;

    Scalar new_norm = 1.0 / slow::sqrt(fi.x * fi.x + fi.y * fi.y + fi.z * fi.z);

    fi.x *= new_norm;
    fi.y *= new_norm;
    fi.z *= new_norm;

    vec3<Scalar> rot_vec = cross(norm, fi);
    rot_vec.x *= slow::sin(phi_half);
    rot_vec.y *= slow::sin(phi_half);
    rot_vec.z *= slow::sin(phi_half);

    quat<Scalar> rot_quat(cos(phi_half), rot_vec);

    quati = rot_quat * quati;

    d_orientation[idx] = quat_to_scalar4(quati);
    }

//! Kernel for applying rotational diffusion to active force vectors on the GPU
/*! \param group_size number of particles
    \param d_index_array stores list to convert group index to global tag
    \param d_pos particle positions on device
    \param d_f_act particle active force unit vector
    \param P position of the ellipsoid constraint
    \param rx radius of the ellipsoid in x direction
    \param ry radius of the ellipsoid in y direction
    \param rz radius of the ellipsoid in z direction
    \param is2D check if simulation is 2D or 3D
    \param rotationConst particle rotational diffusion constant
    \param seed seed for random number generator
*/
__global__ void gpu_compute_active_force_rotational_diffusion_kernel(const unsigned int group_size,
                                                                     unsigned int* d_tag,
                                                                     unsigned int* d_index_array,
                                                                     const Scalar4* d_pos,
                                                                     Scalar4* d_orientation,
                                                                     const Scalar4* d_f_act,
                                                                     const Scalar3 P,
                                                                     const Scalar rx,
                                                                     const Scalar ry,
                                                                     const Scalar rz,
                                                                     bool is2D,
                                                                     const Scalar rotationConst,
                                                                     const uint64_t timestep,
                                                                     const uint16_t seed)
    {
    unsigned int group_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (group_idx >= group_size)
        return;

    unsigned int idx = d_index_array[group_idx];
    Scalar4 posidx = __ldg(d_pos + idx);
    unsigned int type = __scalar_as_int(posidx.w);
    unsigned int ptag = d_tag[group_idx];

    quat<Scalar> quati(__ldg(d_orientation + idx));

    hoomd::RandomGenerator rng(
        hoomd::Seed(hoomd::RNGIdentifier::ActiveForceCompute, timestep, seed),
        hoomd::Counter(ptag));

    if (is2D) // 2D
        {
        Scalar delta_theta; // rotational diffusion angle
        delta_theta = hoomd::NormalDistribution<Scalar>(rotationConst)(rng);
        Scalar theta
            = delta_theta / 2.0; // angle on plane defining orientation of active force vector
        vec3<Scalar> b(0, 0, slow::sin(theta));

        quat<Scalar> rot_quat(slow::cos(theta), b);

        quati = rot_quat * quati;
        d_orientation[idx] = quat_to_scalar4(quati);
        // in 2D there is only one meaningful direction for torque
        }
    else // 3D: Following Stenhammar, Soft Matter, 2014
        {
        if (rx == 0) // if no constraint
            {
            hoomd::SpherePointGenerator<Scalar> unit_vec;
            vec3<Scalar> rand_vec;
            unit_vec(rng, rand_vec);

            Scalar4 fact = __ldg(d_f_act + type);

            vec3<Scalar> f(fact.x, fact.y, fact.z);
            vec3<Scalar> fi = rotate(quati, f);

            vec3<Scalar> aux_vec;
            aux_vec.x = fi.y * rand_vec.z - fi.z * rand_vec.y;
            aux_vec.y = fi.z * rand_vec.x - fi.x * rand_vec.z;
            aux_vec.z = fi.x * rand_vec.y - fi.y * rand_vec.x;
            Scalar aux_vec_mag = 1.0
                                 / slow::sqrt(aux_vec.x * aux_vec.x + aux_vec.y * aux_vec.y
                                              + aux_vec.z * aux_vec.z);
            aux_vec.x *= aux_vec_mag;
            aux_vec.y *= aux_vec_mag;
            aux_vec.z *= aux_vec_mag;

            Scalar delta_theta = hoomd::NormalDistribution<Scalar>(rotationConst)(rng);
            Scalar theta
                = delta_theta / 2.0; // angle on plane defining orientation of active force vector
            quat<Scalar> rot_quat(slow::cos(theta), slow::sin(theta) * aux_vec);

            quati = rot_quat * quati;
            d_orientation[idx].x = quati.s;
            d_orientation[idx].y = quati.v.x;
            d_orientation[idx].z = quati.v.y;
            d_orientation[idx].w = quati.v.z;
            }
        else // if constraint
            {
            EvaluatorConstraintEllipsoid Ellipsoid(P, rx, ry, rz);
            Scalar3 current_pos = make_scalar3(posidx.x, posidx.y, posidx.z);

            Scalar3 norm_scalar3 = Ellipsoid.evalNormal(
                current_pos); // the normal vector to which the particles are confined.
            vec3<Scalar> norm;
            norm = vec3<Scalar>(norm_scalar3);

            Scalar delta_theta = hoomd::NormalDistribution<Scalar>(rotationConst)(rng);
            Scalar theta
                = delta_theta / 2.0; // angle on plane defining orientation of active force vector
            quat<Scalar> rot_quat(slow::cos(theta), slow::sin(theta) * norm);

            quati = rot_quat * quati;
            d_orientation[idx] = quat_to_scalar4(quati);
            }
        }
    }

hipError_t gpu_compute_active_force_set_forces(const unsigned int group_size,
                                               unsigned int* d_index_array,
                                               Scalar4* d_force,
                                               Scalar4* d_torque,
                                               const Scalar4* d_pos,
                                               const Scalar4* d_orientation,
                                               const Scalar4* d_f_act,
                                               const Scalar4* d_t_act,
                                               const Scalar3& P,
                                               const Scalar rx,
                                               const Scalar ry,
                                               const Scalar rz,
                                               const unsigned int N,
                                               unsigned int block_size)
    {
    // setup the grid to run the kernel
    dim3 grid(group_size / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    hipMemset(d_force, 0, sizeof(Scalar4) * N);
    hipLaunchKernelGGL((gpu_compute_active_force_set_forces_kernel),
                       dim3(grid),
                       dim3(threads),
                       0,
                       0,
                       group_size,
                       d_index_array,
                       d_force,
                       d_torque,
                       d_pos,
                       d_orientation,
                       d_f_act,
                       d_t_act,
                       P,
                       rx,
                       ry,
                       rz,
                       N);
    return hipSuccess;
    }

hipError_t gpu_compute_active_force_set_constraints(const unsigned int group_size,
                                                    unsigned int* d_index_array,
                                                    const Scalar4* d_pos,
                                                    Scalar4* d_orientation,
                                                    const Scalar4* d_f_act,
                                                    const Scalar3& P,
                                                    const Scalar rx,
                                                    const Scalar ry,
                                                    const Scalar rz,
                                                    unsigned int block_size)
    {
    // setup the grid to run the kernel
    dim3 grid(group_size / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    hipLaunchKernelGGL((gpu_compute_active_force_set_constraints_kernel),
                       dim3(grid),
                       dim3(threads),
                       0,
                       0,
                       group_size,
                       d_index_array,
                       d_pos,
                       d_orientation,
                       d_f_act,
                       P,
                       rx,
                       ry,
                       rz);
    return hipSuccess;
    }

hipError_t gpu_compute_active_force_rotational_diffusion(const unsigned int group_size,
                                                         unsigned int* d_tag,
                                                         unsigned int* d_index_array,
                                                         const Scalar4* d_pos,
                                                         Scalar4* d_orientation,
                                                         const Scalar4* d_f_act,
                                                         const Scalar3& P,
                                                         const Scalar rx,
                                                         const Scalar ry,
                                                         const Scalar rz,
                                                         bool is2D,
                                                         const Scalar rotationConst,
                                                         const uint64_t timestep,
                                                         const uint16_t seed,
                                                         unsigned int block_size)
    {
    // setup the grid to run the kernel
    dim3 grid(group_size / block_size + 1, 1, 1);
    dim3 threads(block_size, 1, 1);

    // run the kernel
    hipLaunchKernelGGL((gpu_compute_active_force_rotational_diffusion_kernel),
                       dim3(grid),
                       dim3(threads),
                       0,
                       0,
                       group_size,
                       d_tag,
                       d_index_array,
                       d_pos,
                       d_orientation,
                       d_f_act,
                       P,
                       rx,
                       ry,
                       rz,
                       is2D,
                       rotationConst,
                       timestep,
                       seed);
    return hipSuccess;
    }
