#include "hip/hip_runtime.h"
// Copyright (c) 2009-2018 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.


// Maintainer: jglaser

#include "ParticleData.cuh"
#include "ParticleGroup.cuh"

#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

/*! \file ParticleGroup.cu
    \brief Contains GPU kernel code used by ParticleGroup
*/

//! GPU kernel to translate between global and local membership lookup table
__global__ void gpu_rebuild_index_list_kernel(unsigned int N,
                                              unsigned int *d_tag,
                                              unsigned int *d_is_member_tag,
                                              unsigned int *d_is_member)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N) return;

    unsigned int tag = d_tag[idx];

    d_is_member[idx] = d_is_member_tag[tag];
    }

__global__ void gpu_scatter_member_indices(unsigned int N,
    const unsigned int *d_scan,
    const unsigned int *d_is_member,
    unsigned *d_member_idx)
    {
    unsigned int idx = blockIdx.x*blockDim.x+threadIdx.x;

    if (idx >= N) return;

    if (d_is_member[idx])
        d_member_idx[d_scan[idx]] = idx;
    }

//! GPU method for rebuilding the index list of a ParticleGroup
/*! \param N number of local particles
    \param d_is_member_tag Global lookup table for tag -> group membership
    \param d_is_member Array of membership flags
    \param d_member_idx Array of member indices
    \param d_tag Array of tags
    \param num_local_members Number of members on the local processor (return value)
*/
hipError_t gpu_rebuild_index_list(unsigned int N,
                                   unsigned int *d_is_member_tag,
                                   unsigned int *d_is_member,
                                   unsigned int *d_member_idx,
                                   unsigned int *d_tag,
                                   unsigned int &num_local_members,
                                   unsigned int *d_tmp,
                                   const CachedAllocator& alloc)
    {
    assert(d_is_member);
    assert(d_is_member_tag);
    assert(d_member_idx);
    assert(d_tag);

    unsigned int block_size = 512;
    unsigned int n_blocks = N/block_size + 1;

    gpu_rebuild_index_list_kernel<<<n_blocks,block_size>>>(N,
                                                         d_tag,
                                                         d_is_member_tag,
                                                         d_is_member);

    // compute member_idx offsets
    thrust::device_ptr<unsigned int> is_member(d_is_member);
    thrust::device_ptr<unsigned int> tmp(d_tmp);
    thrust::exclusive_scan(thrust::cuda::par(alloc),
        is_member,
        is_member + N,
        tmp);

    num_local_members = thrust::reduce(thrust::cuda::par(alloc),
        is_member,
        is_member + N);

    // fill member_idx array
    gpu_scatter_member_indices<<<n_blocks, block_size>>>(N, d_tmp, d_is_member, d_member_idx);

    return hipSuccess;
    }
