#include "hip/hip_runtime.h"
// Copyright (c) 2009-2016 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "IntegratorHPMCMonoGPU.cuh"

#include "Moves.h"
#include "ShapeSphere.h"
#include "ShapeConvexPolygon.h"
#include "ShapeSpheropolygon.h"
#include "ShapePolyhedron.h"
#include "ShapeConvexPolyhedron.h"
#include "ShapeSpheropolyhedron.h"
#include "ShapeSimplePolygon.h"
#include "ShapeEllipsoid.h"
#include "ShapeFacetedSphere.h"
#include "ShapeSphinx.h"
#include "ShapeUnion.h"
#include "hoomd/TextureTools.h"

#include <stdio.h>

namespace hpmc
{

namespace detail
{

/*! \file IntegratorHPMCMonoGPU.cu
    \brief Definition of CUDA kernels and drivers for IntegratorHPMCMono
*/

//! Texture for reading postype
scalar4_tex_t postype_tex;
//! Texture for reading orientation
scalar4_tex_t orientation_tex;
//! Texture for reading cell index data
texture<unsigned int, 1, hipReadModeElementType> cell_idx_tex;

//! Compute the cell that a particle sits in
__device__ inline unsigned int computeParticleCell(const Scalar3& p,
                                                   const BoxDim& box,
                                                   const Scalar3& ghost_width,
                                                   const uint3& cell_dim,
                                                   const Index3D& ci)
    {
    // find the bin each particle belongs in
    Scalar3 f = box.makeFraction(p,ghost_width);
    uchar3 periodic = box.getPeriodic();
    int ib = (unsigned int)(f.x * cell_dim.x);
    int jb = (unsigned int)(f.y * cell_dim.y);
    int kb = (unsigned int)(f.z * cell_dim.z);

    // need to handle the case where the particle is exactly at the box hi
    if (ib == (int)cell_dim.x && periodic.x)
        ib = 0;
    if (jb == (int)cell_dim.y && periodic.y)
        jb = 0;
    if (kb == (int)cell_dim.z && periodic.z)
        kb = 0;

    // identify the bin
    if (f.x >= Scalar(0.0) && f.x < Scalar(1.0) && f.y >= Scalar(0.0) && f.y < Scalar(1.0) && f.z >= Scalar(0.0) && f.z < Scalar(1.0))
        return ci(ib,jb,kb);
    else
        return 0xffffffff;
    }

//! Kernel to generate expanded cells
/*! \param d_excell_idx Output array to list the particle indices in the expanded cells
    \param d_excell_size Output array to list the number of particles in each expanded cell
    \param excli Indexer for the expanded cells
    \param d_cell_idx Particle indices in the normal cells
    \param d_cell_size Number of particles in each cell
    \param d_cell_adj Cell adjacency list
    \param ci Cell indexer
    \param cli Cell list indexer
    \param cadji Cell adjacency indexer

    gpu_hpmc_excell_kernel executes one thread per cell. It gathers the particle indices from all neighboring cells
    into the output expanded cell.
*/
__global__ void gpu_hpmc_excell_kernel(unsigned int *d_excell_idx,
                                       unsigned int *d_excell_size,
                                       const Index2D excli,
                                       const unsigned int *d_cell_idx,
                                       const unsigned int *d_cell_size,
                                       const unsigned int *d_cell_adj,
                                       const Index3D ci,
                                       const Index2D cli,
                                       const Index2D cadji)
    {
    // compute the output cell
    unsigned int my_cell = 0;
    if (gridDim.y > 1)
        {
        // if gridDim.y > 1, then the fermi workaround is in place, index blocks on a 2D grid
        my_cell = (blockIdx.x + blockIdx.y * 65535) * blockDim.x + threadIdx.x;
        }
    else
        {
        my_cell = blockDim.x * blockIdx.x + threadIdx.x;
        }

    if (my_cell >= ci.getNumElements())
        return;

    unsigned int my_cell_size = 0;

    // loop over neighboring cells and build up the expanded cell list
    for (unsigned int offset = 0; offset < cadji.getW(); offset++)
        {
        unsigned int neigh_cell = d_cell_adj[cadji(offset, my_cell)];
        unsigned int neigh_cell_size = d_cell_size[neigh_cell];

        for (unsigned int k = 0; k < neigh_cell_size; k++)
            {
            // read in the index of the new particle to add to our cell
            unsigned int new_idx = tex1Dfetch(cell_idx_tex, cli(k, neigh_cell));
            d_excell_idx[excli(my_cell_size, my_cell)] = new_idx;
            my_cell_size++;
            }
        }

    // write out the final size
    d_excell_size[my_cell] = my_cell_size;
    }

//! Kernel driver for gpu_hpmc_excell_kernel()
hipError_t gpu_hpmc_excell(unsigned int *d_excell_idx,
                            unsigned int *d_excell_size,
                            const Index2D& excli,
                            const unsigned int *d_cell_idx,
                            const unsigned int *d_cell_size,
                            const unsigned int *d_cell_adj,
                            const Index3D& ci,
                            const Index2D& cli,
                            const Index2D& cadji,
                            const unsigned int block_size)
    {
    assert(d_excell_idx);
    assert(d_excell_size);
    assert(d_cell_idx);
    assert(d_cell_size);
    assert(d_cell_adj);

    // determine the maximum block size and clamp the input block size down
    static int max_block_size = -1;
    static int sm = -1;
    if (max_block_size == -1)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(gpu_hpmc_excell_kernel));
        max_block_size = attr.maxThreadsPerBlock;
        sm = attr.binaryVersion;
        }

    // setup the grid to run the kernel
    dim3 threads(min(block_size, (unsigned int)max_block_size), 1, 1);
    dim3 grid(ci.getNumElements() / block_size + 1, 1, 1);

    // hack to enable grids of more than 65k blocks
    if (sm < 30 && grid.x > 65535)
        {
        grid.y = grid.x / 65535 + 1;
        grid.x = 65535;
        }

    // bind the textures
    cell_idx_tex.normalized = false;
    cell_idx_tex.filterMode = hipFilterModePoint;
    hipError_t error = hipBindTexture(0, cell_idx_tex, d_cell_idx, sizeof(unsigned int)*cli.getNumElements());
    if (error != hipSuccess)
        return error;

    gpu_hpmc_excell_kernel<<<grid, threads>>>(d_excell_idx,
                                              d_excell_size,
                                              excli,
                                              d_cell_idx,
                                              d_cell_size,
                                              d_cell_adj,
                                              ci,
                                              cli,
                                              cadji);

    return hipSuccess;
    }


//! HPMC  update kernel
/*! \param d_postype Particle positions and types by index
    \param d_orientation Particle orientation
    \param d_counters Acceptance counters to increment
    \param d_cell_idx Particle index stored in the cell list
    \param d_cell_size The size of each cell
    \param d_excell_idx Indices of particles in extended cells
    \param d_excell_size Number of particles in each extended cell
    \param ci Cell indexer
    \param cli Cell list indexer
    \param excli Extended cell list indexer
    \param cell_dim Dimensions of the cell list
    \param ghost_width Width of the ghost layer
    \param d_cell_set List of active cells
    \param n_active_cells Number of active cells
    \param N number of particles
    \param num_types Number of particle types
    \param seed User chosen random number seed
    \param d_d Array of maximum move displacements
    \param d_a Array of rotation move sizes
    \param d_check_overlaps Interaction matrix
    \parma overlap_idx Indexer into interaction matrix
    \param move_ratio Ratio of translation moves to rotation moves
    \param timestep Current timestep of the simulation
    \param dim Dimension of the simulation box
    \param box Simulation box
    \param select Current index within the loop over nselect selections (for RNG generation)
    \param ghost_fraction Width of the inactive layer in MPI domain decomposition simulations
    \param domain_decomposition True if executing with domain decomposition
    \param d_params Per-type shape parameters

    MPMC in its published form has a severe limit on the number of parallel threads in 3D. This implementation launches
    group_size threads per cell (1,2,4,8,16,32). Each thread in the group performs the same trial move on the same
    particle, and then checks for overlaps against different particles from the extended cell list. The entire extended
    cell list is covered in a batched loop. The group_size is autotuned to find the fastest performance. Smaller systems
    tend to run fastest with a large group_size due to the increased parallelism. Larger systems tend to run faster
    at smaller group_sizes because they already have the parallelism from the system size - however, even the largest
    systems benefit from group_size > 1 on K20. Shared memory is used to set an overlap flag to 1 if any of the threads
    in the group detect an overlap. After all checks are complete, the master thread in the group applies the trial move
    update if accepted.

    No __synchtreads is needed after the overlap checks because the group_size is always chosen to be a power of 2 and
    smaller than the warp size. Only a __threadfence_block() is needed to ensure memory consistency.

    Move stats are tallied in local memory, then totaled in shared memory at the end and finally a single thread in the
    block runs an atomicAdd on global memory to get the system wide total. This isn't as good as a reduction, but it
    is only a tiny fraction of the compute time.

    In order to simplify indexing and boundary checks, a list of active cells is determined on the host and passed into
    the kernel. That way, only a linear indexing of threads is needed to handle any geometry of active cells.

    Heavily divergent warps are avoided by pre-building a list of all particles in the neighboring region of any given
    cell. Otherwise, extremely non-uniform cell lengths (i.e. avg 1, max 4) don't cause massive performance degradation.

    **Indexing**
        - threadIdx.z indexes the current group in the block
        - threadIdx.x is the offset within the current group
        - blockIdx.x runs enough blocks so that all active cells are covered

    **Possible enhancements**
        - Use __ldg and not tex1Dfetch on sm35

    \ingroup hpmc_kernels
*/
template< class Shape >
__global__ void gpu_hpmc_mpmc_kernel(Scalar4 *d_postype,
                                     Scalar4 *d_orientation,
                                     hpmc_counters_t *d_counters,
                                     const unsigned int *d_cell_idx,
                                     const unsigned int *d_cell_size,
                                     const unsigned int *d_excell_idx,
                                     const unsigned int *d_excell_size,
                                     const Index3D ci,
                                     const Index2D cli,
                                     const Index2D excli,
                                     const uint3 cell_dim,
                                     const Scalar3 ghost_width,
                                     const unsigned int *d_cell_set,
                                     const unsigned int n_active_cells,
                                     const unsigned int N,
                                     const unsigned int num_types,
                                     const unsigned int seed,
                                     const Scalar* d_d,
                                     const Scalar* d_a,
                                     const unsigned int *d_check_overlaps,
                                     const Index2D overlap_idx,
                                     const unsigned int move_ratio,
                                     const unsigned int timestep,
                                     const unsigned int dim,
                                     const BoxDim box,
                                     const unsigned int select,
                                     const Scalar3 ghost_fraction,
                                     const bool domain_decomposition,
                                     unsigned int *d_active_cell_ptl_idx,
                                     unsigned int *d_active_cell_accept,
                                     unsigned int *d_active_cell_move_type_translate,
                                     const typename Shape::param_type *d_params,
                                     unsigned int max_queue_size)
    {
    // flags to tell what type of thread we are
    bool active = true;
    unsigned int group;
    unsigned int offset;
    unsigned int group_size;
    bool master;
    unsigned int n_groups;

    if (Shape::isParallel())
        {
        // use 3d thread block layout
        group = threadIdx.z;
        offset = threadIdx.y;
        group_size = blockDim.y;
        master = (offset == 0 && threadIdx.x == 0);
        n_groups = blockDim.z;
        }
    else
        {
        group = threadIdx.y;
        offset = threadIdx.x;
        group_size = blockDim.x;
        master = (offset == 0);
        n_groups = blockDim.y;
        }

    unsigned int err_count = 0;

    // shared arrays for per type pair parameters
    __shared__ unsigned int s_translate_accept_count;
    __shared__ unsigned int s_translate_reject_count;
    __shared__ unsigned int s_rotate_accept_count;
    __shared__ unsigned int s_rotate_reject_count;
    __shared__ unsigned int s_overlap_checks;
    __shared__ unsigned int s_overlap_err_count;

    __shared__ unsigned int s_queue_size;
    __shared__ unsigned int s_still_searching;

    // load the per type pair parameters into shared memory
    extern __shared__ char s_data[];
    typename Shape::param_type *s_params = (typename Shape::param_type *)(&s_data[0]);
    Scalar4 *s_orientation_group = (Scalar4*)(s_params + num_types);
    Scalar3 *s_pos_group = (Scalar3*)(s_orientation_group + n_groups);
    Scalar *s_d = (Scalar *)(s_pos_group + n_groups);
    Scalar *s_a = (Scalar *)(s_d + num_types);
    unsigned int *s_check_overlaps = (unsigned int *) (s_a + num_types);
    unsigned int *s_queue_j =   (unsigned int*)(s_check_overlaps + overlap_idx.getNumElements());
    unsigned int *s_overlap =   (unsigned int*)(s_queue_j + max_queue_size);
    unsigned int *s_queue_gid = (unsigned int*)(s_overlap + n_groups);
    unsigned int *s_type_group = (unsigned int*)(s_queue_gid + max_queue_size);

    // copy over parameters one int per thread for fast loads
        {
        unsigned int tidx = threadIdx.x+blockDim.x*threadIdx.y + blockDim.x*blockDim.y*threadIdx.z;
        unsigned int block_size = blockDim.x*blockDim.y*blockDim.z;
        unsigned int param_size = num_types*sizeof(typename Shape::param_type) / sizeof(int);

        for (unsigned int cur_offset = 0; cur_offset < param_size; cur_offset += block_size)
            {
            if (cur_offset + tidx < param_size)
                {
                ((int *)s_params)[cur_offset + tidx] = ((int *)d_params)[cur_offset + tidx];
                }
            }

        for (unsigned int cur_offset = 0; cur_offset < num_types; cur_offset += block_size)
            {
            if (cur_offset + tidx < num_types)
                {
                s_a[cur_offset + tidx] = d_a[cur_offset + tidx];
                s_d[cur_offset + tidx] = d_d[cur_offset + tidx];
                }
            }

        unsigned int ntyppairs = overlap_idx.getNumElements();

        for (unsigned int cur_offset = 0; cur_offset < ntyppairs; cur_offset += block_size)
            {
            if (cur_offset + tidx < ntyppairs)
                {
                s_check_overlaps[cur_offset + tidx] = d_check_overlaps[cur_offset + tidx];
                }
            }
        }

    // initialize the shared memory array for communicating overlaps
    if (master && group == 0)
        {
        s_translate_accept_count = 0;
        s_translate_reject_count = 0;
        s_rotate_accept_count = 0;
        s_rotate_reject_count = 0;
        s_overlap_checks = 0;
        s_overlap_err_count = 0;
        s_queue_size = 0;
        s_still_searching = 1;
        }
    if (master)
        {
        s_overlap[group] = 0;
        }

    // identify the active cell that this thread handles
    unsigned int active_cell_idx = 0;
    if (gridDim.y > 1)
        {
        // if gridDim.y > 1, then the fermi workaround is in place, index blocks on a 2D grid
        active_cell_idx = (blockIdx.x + blockIdx.y * 65535) * n_groups + group;
        }
    else
        {
        active_cell_idx = blockIdx.x * n_groups + group;
        }


    // this thread is inactive if it indexes past the end of the active cell list
    if (active_cell_idx >= n_active_cells)
        active = false;

    // pull in the index of our cell
    unsigned int my_cell = 0;
    unsigned int my_cell_size = 0;
    if (active)
        {
        my_cell = d_cell_set[active_cell_idx];
        my_cell_size = d_cell_size[my_cell];
        }

    // need to deactivate if there are no particles in this cell
    if (my_cell_size == 0)
        active = false;

    __syncthreads();

    // initial implementation just moves one particle per cell (nselect=1).
    // these variables are ugly, but needed to get the updated quantities outside of the scope
    unsigned int i;
    unsigned int overlap_checks = 0;
    bool move_type_translate = false;
    bool move_active = true;
    int ignore_stats = 0;

    if (active)
        {
        // one RNG per cell
        SaruGPU rng(my_cell, seed+select, timestep);

        // select one of the particles randomly from the cell
        unsigned int my_cell_offset = rand_select(rng, my_cell_size-1);
        i = tex1Dfetch(cell_idx_tex, cli(my_cell_offset, my_cell));

        // read in the position and orientation of our particle.
        Scalar4 postype_i = texFetchScalar4(d_postype, postype_tex, i);
        Scalar4 orientation_i = make_scalar4(1,0,0,0);

        unsigned int typ_i = __scalar_as_int(postype_i.w);
        Shape shape_i(quat<Scalar>(orientation_i), s_params[typ_i]);

        if (shape_i.hasOrientation())
            orientation_i = texFetchScalar4(d_orientation, orientation_tex, i);

        shape_i.orientation = quat<Scalar>(orientation_i);

        // if this looks funny, that is because it is. Using ignore_stats as a bool setting ignore_stats = ...
        // causes a compiler bug.
        if (shape_i.ignoreStatistics())
            ignore_stats = 1;

        vec3<Scalar> pos_i = vec3<Scalar>(postype_i);

        // for domain decomposition simulations, we need to leave all particles in the inactive region alone
        // in order to avoid even more divergence, this is done by setting the move_active flag
        // overlap checks are still processed, but the final move acceptance will be skipped
        if (domain_decomposition && !isActive(make_scalar3(postype_i.x, postype_i.y, postype_i.z), box, ghost_fraction))
            move_active = false;

        // make the move
        unsigned int move_type_select = rng.u32() & 0xffff;
        move_type_translate = !shape_i.hasOrientation() || (move_type_select < move_ratio);

        if (move_type_translate)
            {
            move_translate(pos_i, rng, s_d[typ_i], dim);

            // need to reject any move that puts the particle in the inactive region
            if (domain_decomposition && !isActive(vec_to_scalar3(pos_i), box, ghost_fraction))
                move_active = false;
            }
        else
            {
            move_rotate(shape_i.orientation, rng, s_a[typ_i], dim);
            }

        // stash the trial move in shared memory so that other threads in this block can process overlap checks
        if (master)
            {
            s_pos_group[group] = make_scalar3(pos_i.x, pos_i.y, pos_i.z);
            s_type_group[group] = typ_i;
            s_orientation_group[group] = quat_to_scalar4(shape_i.orientation);
            }
        }

    // sync so that s_postype_group and s_orientation are available before other threads might process overlap checks
    __syncthreads();

    // counters to track progress through the loop over potential neighbors
    unsigned int excell_size;
    unsigned int k = offset;
    if (active)
        {
        excell_size = d_excell_size[my_cell];
        overlap_checks += excell_size;
        }

    // loop while still searching
    while (s_still_searching)
        {
        // stage 1, fill the queue.
        // loop through particles in the excell list and add them to the queue if they pass the circumsphere check

        // active threads add to the queue
        if (active)
            {
            // prefetch j
            unsigned int j, next_j = 0;
            if (k < excell_size)
                {
                #if (__CUDA_ARCH__ > 300)
                next_j = __ldg(&d_excell_idx[excli(k, my_cell)]);
                #else
                next_j = d_excell_idx[excli(k, my_cell)];
                #endif
                }

            // add to the queue as long as the queue is not full, and we have not yet reached the end of our own list
            // and as long as no overlaps have been found
            while (!s_overlap[group] && s_queue_size < max_queue_size && k < excell_size)
                {
                if (k < excell_size)
                    {
                    Scalar4 postype_j;
                    Scalar4 orientation_j;
                    vec3<Scalar> r_ij;

                    // build some shapes, but we only need them to get diameters, so don't load orientations
                    // build shape i from shared memory
                    Scalar3 pos_i = s_pos_group[group];
                    Shape shape_i(quat<Scalar>(), s_params[s_type_group[group]]);

                    // prefetch next j
                    k += group_size;
                    j = next_j;

                    if (k < excell_size)
                        {
                        #if (__CUDA_ARCH__ > 300)
                        next_j = __ldg(&d_excell_idx[excli(k, my_cell)]);
                        #else
                        next_j = d_excell_idx[excli(k, my_cell)];
                        #endif
                        }

                    // read in position, and orientation of neighboring particle
                    postype_j = texFetchScalar4(d_postype, postype_tex, j);
                    Shape shape_j(quat<Scalar>(orientation_j), s_params[__scalar_as_int(postype_j.w)]);

                    // put particle j into the coordinate system of particle i
                    r_ij = vec3<Scalar>(postype_j) - vec3<Scalar>(pos_i);
                    r_ij = vec3<Scalar>(box.minImage(vec_to_scalar3(r_ij)));

                    // test circumsphere overlap
                    OverlapReal rsq = dot(r_ij,r_ij);
                    OverlapReal DaDb = shape_i.getCircumsphereDiameter() + shape_j.getCircumsphereDiameter();

                    if (i != j && rsq*OverlapReal(4.0) <= DaDb * DaDb)
                        {
                        // add this particle to the queue
                        unsigned int insert_point = atomicAdd(&s_queue_size, 1);

                        if (insert_point < max_queue_size)
                            {
                            s_queue_gid[insert_point] = group;
                            s_queue_j[insert_point] = j;
                            }
                        else
                            {
                            // or back up if the queue is already full
                            // we will recheck and insert this on the next time through
                            k -= group_size;
                            }
                        }

                    } // end if k < excell_size
                } // end while (s_queue_size < max_queue_size && k < excell_size)
            } // end if active

        // sync to make sure all threads in the block are caught up
        __syncthreads();

        // when we get here, all threads have either finished their list, or encountered a full queue
        // either way, it is time to process overlaps
        // need to clear the still searching flag and sync first
        if (master && group == 0)
            s_still_searching = 0;

        unsigned int tidx_1d = threadIdx.x+blockDim.x*threadIdx.y + blockDim.x*blockDim.y*threadIdx.z;

        // max_queue_size is always <= block size, so we just need an if here
        if (tidx_1d < min(s_queue_size, max_queue_size))
            {
            // need to extract the overlap check to perform out of the shared mem queue
            unsigned int check_group = s_queue_gid[tidx_1d];
            unsigned int check_j = s_queue_j[tidx_1d];
            Scalar4 postype_j;
            Scalar4 orientation_j;
            vec3<Scalar> r_ij;

            // build shape i from shared memory
            Scalar3 pos_i = s_pos_group[check_group];
            unsigned int type_i = s_type_group[check_group];
            Shape shape_i(quat<Scalar>(s_orientation_group[check_group]), s_params[type_i]);

            // build shape j from global memory
            postype_j = texFetchScalar4(d_postype, postype_tex, check_j);
            orientation_j = make_scalar4(1,0,0,0);
            unsigned int type_j = __scalar_as_int(postype_j.w);
            Shape shape_j(quat<Scalar>(orientation_j), s_params[type_j]);
            if (shape_j.hasOrientation())
                shape_j.orientation = quat<Scalar>(texFetchScalar4(d_orientation, orientation_tex, check_j));

            // put particle j into the coordinate system of particle i
            r_ij = vec3<Scalar>(postype_j) - vec3<Scalar>(pos_i);
            r_ij = vec3<Scalar>(box.minImage(vec_to_scalar3(r_ij)));

            if (s_check_overlaps[overlap_idx(type_i, type_j)] && test_overlap(r_ij, shape_i, shape_j, err_count))
                {
                atomicAdd(&s_overlap[check_group], 1);
                }
            }

        // threads that need to do more looking set the still_searching flag
        __syncthreads();
        if (master && group == 0)
            s_queue_size = 0;

        if (active && !s_overlap[group] && k < excell_size)
            atomicAdd(&s_still_searching, 1);
        __syncthreads();

        } // end while (s_still_searching)

    // update the data if accepted
    if (master)
        {
        if (active && move_active)
            {
            // first need to check if the particle remains in its cell
            Scalar3 xnew_i = s_pos_group[group];
            unsigned int new_cell = computeParticleCell(xnew_i, box, ghost_width, cell_dim, ci);
            bool accepted=true;
            if (s_overlap[group])
                accepted=false;
            if (new_cell != my_cell)
                accepted=false;

            if (accepted)
                {
                // write out the updated position and orientation
                d_postype[i] = make_scalar4(xnew_i.x, xnew_i.y, xnew_i.z, __int_as_scalar(s_type_group[group]));
                d_orientation[i] = s_orientation_group[group];
                }

            if (d_active_cell_accept)
                {
                // store particle index
                d_active_cell_ptl_idx[active_cell_idx] = i;
                }

            if (d_active_cell_accept)
                {
                // store accept flag
                d_active_cell_accept[active_cell_idx] = accepted ? 1 : 0;
                }

            if (d_active_cell_move_type_translate)
                {
                // store move type
                d_active_cell_move_type_translate[active_cell_idx] = move_type_translate ? 1 : 0;
                }

            // if an auxillary array was provided, defer writing out statistics
            if (d_active_cell_ptl_idx)
                {
                ignore_stats = 1;
                }

            if (!ignore_stats && accepted && move_type_translate)
                atomicAdd(&s_translate_accept_count, 1);
            if (!ignore_stats && accepted && !move_type_translate)
                atomicAdd(&s_rotate_accept_count, 1);
            if (!ignore_stats && !accepted && move_type_translate)
                atomicAdd(&s_translate_reject_count, 1);
            if (!ignore_stats && !accepted && !move_type_translate)
                atomicAdd(&s_rotate_reject_count, 1);
            }
        else // active && move_active
            {
            if (d_active_cell_ptl_idx && active_cell_idx < n_active_cells)
                {
                // indicate that no particle was selected
                d_active_cell_ptl_idx[active_cell_idx] = UINT_MAX;
                }
            }

        // count the overlap checks
        atomicAdd(&s_overlap_checks, overlap_checks);
        }

    if (err_count > 0)
        atomicAdd(&s_overlap_err_count, err_count);

    __syncthreads();

    // final tally into global mem
    if (master && group == 0)
        {
        atomicAdd(&d_counters->translate_accept_count, s_translate_accept_count);
        atomicAdd(&d_counters->translate_reject_count, s_translate_reject_count);
        atomicAdd(&d_counters->rotate_accept_count, s_rotate_accept_count);
        atomicAdd(&d_counters->rotate_reject_count, s_rotate_reject_count);
        atomicAdd(&d_counters->overlap_checks, s_overlap_checks);
        atomicAdd(&d_counters->overlap_err_count, s_overlap_err_count);
        }
    }

//! Kernel for grid shift
/*! \param d_postype postype of each particle
    \param d_image Image flags for each particle
    \param N number of particles
    \param box Simulation box
    \param shift Vector by which to translate the particles

    Shift all the particles by a given vector.

    \ingroup hpmc_kernels
*/
__global__ void gpu_hpmc_shift_kernel(Scalar4 *d_postype,
                                      int3 *d_image,
                                      const unsigned int N,
                                      const BoxDim box,
                                      const Scalar3 shift)
    {
    // identify the active cell that this thread handles
    unsigned int my_pidx = blockIdx.x * blockDim.x + threadIdx.x;

    // this thread is inactive if it indexes past the end of the particle list
    if (my_pidx >= N)
        return;

    // pull in the current position
    Scalar4 postype = d_postype[my_pidx];

    // shift the position
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    pos += shift;

    // wrap the particle back into the box
    int3 image = d_image[my_pidx];
    box.wrap(pos, image);

    // write out the new position and orientation
    d_postype[my_pidx] = make_scalar4(pos.x, pos.y, pos.z, postype.w);
    d_image[my_pidx] = image;
    }


//! Kernel driver for gpu_update_hpmc_kernel()
/*! \param args Bundled arguments
    \param d_params Per-type shape parameters
    \returns Error codes generated by any CUDA calls, or hipSuccess when there is no error

    This templatized method is the kernel driver for HPMC update of any shape. It is instantiated for every shape at the
    bottom of this file.

    \ingroup hpmc_kernels
*/
template< class Shape >
hipError_t gpu_hpmc_update(const hpmc_args_t& args, const typename Shape::param_type *d_params)
    {
    assert(args.d_postype);
    assert(args.d_orientation);
    assert(args.d_counters);
    assert(args.d_cell_idx);
    assert(args.d_cell_size);
    assert(args.d_excell_idx);
    assert(args.d_excell_size);
    assert(args.d_cell_set);
    assert(args.d_d);
    assert(args.d_a);
    assert(args.d_check_overlaps);
    assert(args.group_size >= 1);
    assert(args.block_size%(args.stride*args.group_size)==0);

    // determine the maximum block size and clamp the input block size down
    static int max_block_size = -1;
    static int sm = -1;
    static hipFuncAttributes attr;
    if (max_block_size == -1)
        {
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(gpu_hpmc_mpmc_kernel<Shape>));
        max_block_size = attr.maxThreadsPerBlock;
        sm = attr.binaryVersion;
        }

    // might need to modify group_size to make the kernel runnable
    unsigned int group_size = args.group_size;

    // choose a block size based on the max block size by regs (max_block_size) and include dynamic shared memory usage
    unsigned int block_size = min(args.block_size, (unsigned int)max_block_size);

    // the new block size might not be a multiple of group size, decrease group size until it is
    group_size = args.group_size;

    while ((block_size%(args.stride*group_size)) != 0)
        {
        group_size--;
        }

    unsigned int n_groups = block_size / group_size / args.stride;
    unsigned int shared_bytes = n_groups * (sizeof(unsigned int)*2 + sizeof(Scalar4) + sizeof(Scalar3)) +
                                block_size*(sizeof(unsigned int) + sizeof(unsigned int)) +
                                args.num_types * (sizeof(typename Shape::param_type) + 2*sizeof(Scalar)) +
                                args.overlap_idx.getNumElements() * sizeof(unsigned int);

    if (args.num_types * (sizeof(typename Shape::param_type) + 2*sizeof(Scalar)) >= args.devprop.sharedMemPerBlock)
        throw std::runtime_error("Insufficient shared memory for HPMC kernel: reduce number of particle types or size of shape parameters");

    while (shared_bytes + attr.sharedSizeBytes >= args.devprop.sharedMemPerBlock)
        {
        block_size -= args.devprop.warpSize;
        if (block_size == 0)
            throw std::runtime_error("Insufficient shared memory for HPMC kernel");

        // the new block size might not be a multiple of group size, decrease group size until it is
        group_size = args.group_size;

        while ((block_size%(args.stride*group_size)) != 0)
            {
            group_size--;
            }

        n_groups = block_size / group_size / args.stride;
        shared_bytes = n_groups * (sizeof(unsigned int)*2 + sizeof(Scalar4) + sizeof(Scalar3)) +
                       block_size*(sizeof(unsigned int) + sizeof(unsigned int)) +
                       args.num_types * (sizeof(typename Shape::param_type) + 2*sizeof(Scalar)) +
                       args.overlap_idx.getNumElements() * sizeof(unsigned int);
        }

    // setup the grid to run the kernel
    dim3 threads;
    if (Shape::isParallel())
        {
        // use three-dimensional thread-layout with blockDim.z < 64
        threads = dim3(args.stride, group_size, n_groups);
        }
    else
        {
        threads = dim3(group_size, n_groups,1);
        }

    dim3 grid( args.n_active_cells / n_groups + 1, 1, 1);

    // hack to enable grids of more than 65k blocks
    if (sm < 30 && grid.x > 65535)
        {
        grid.y = grid.x / 65535 + 1;
        grid.x = 65535;
        }

    // bind the textures
    postype_tex.normalized = false;
    postype_tex.filterMode = hipFilterModePoint;
    hipError_t error = hipBindTexture(0, postype_tex, args.d_postype, sizeof(Scalar4)*args.max_n);
    if (error != hipSuccess)
        return error;

    if (args.has_orientation)
        {
        orientation_tex.normalized = false;
        orientation_tex.filterMode = hipFilterModePoint;
        error = hipBindTexture(0, orientation_tex, args.d_orientation, sizeof(Scalar4)*args.max_n);
        if (error != hipSuccess)
            return error;
        }

    cell_idx_tex.normalized = false;
    cell_idx_tex.filterMode = hipFilterModePoint;
    error = hipBindTexture(0, cell_idx_tex, args.d_cell_idx, sizeof(Scalar4)*args.cli.getNumElements());
    if (error != hipSuccess)
        return error;

    gpu_hpmc_mpmc_kernel<Shape><<<grid, threads, shared_bytes>>>(args.d_postype,
                                                                 args.d_orientation,
                                                                 args.d_counters,
                                                                 args.d_cell_idx,
                                                                 args.d_cell_size,
                                                                 args.d_excell_idx,
                                                                 args.d_excell_size,
                                                                 args.ci,
                                                                 args.cli,
                                                                 args.excli,
                                                                 args.cell_dim,
                                                                 args.ghost_width,
                                                                 args.d_cell_set,
                                                                 args.n_active_cells,
                                                                 args.N,
                                                                 args.num_types,
                                                                 args.seed,
                                                                 args.d_d,
                                                                 args.d_a,
                                                                 args.d_check_overlaps,
                                                                 args.overlap_idx,
                                                                 args.move_ratio,
                                                                 args.timestep,
                                                                 args.dim,
                                                                 args.box,
                                                                 args.select,
                                                                 args.ghost_fraction,
                                                                 args.domain_decomposition,
                                                                 args.d_active_cell_ptl_idx,
                                                                 args.d_active_cell_accept,
                                                                 args.d_active_cell_move_type_translate,
                                                                 d_params,
                                                                 block_size);

    return hipSuccess;
    }

//! Kernel driver for gpu_hpmc_shift_kernel()
hipError_t gpu_hpmc_shift(Scalar4 *d_postype,
                           int3 *d_image,
                           const unsigned int N,
                           const BoxDim& box,
                           const Scalar3 shift,
                           const unsigned int block_size)
    {
    assert(d_postype);
    assert(d_image);

    // setup the grid to run the kernel
    dim3 threads_shift(block_size, 1, 1);
    dim3 grid_shift(N / block_size + 1, 1, 1);

    gpu_hpmc_shift_kernel<<<grid_shift, threads_shift>>>(d_postype,
                                                         d_image,
                                                         N,
                                                         box,
                                                         shift);

    return hipSuccess;
    }

// Instantiate shape templates

//! HPMC update for ShapeSphere
template hipError_t gpu_hpmc_update<ShapeSphere>(const hpmc_args_t& args,
                                                  const typename ShapeSphere::param_type *d_params);

//! HPMC update for ShapeConvexPolygon
template hipError_t gpu_hpmc_update<ShapeConvexPolygon>(const hpmc_args_t& args,
                                                         const typename ShapeConvexPolygon::param_type *d_params);

//! HPMC update for ShapePolyhedron
template hipError_t gpu_hpmc_update<ShapePolyhedron>(const hpmc_args_t& args,
                                                      const typename ShapePolyhedron::param_type *d_params);

//! HPMC update for ShapeConvexPolyhedron
template hipError_t gpu_hpmc_update<ShapeConvexPolyhedron<8> >(const hpmc_args_t& args,
                                                            const typename ShapeConvexPolyhedron<8> ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeConvexPolyhedron<16> >(const hpmc_args_t& args,
                                                            const typename ShapeConvexPolyhedron<16> ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeConvexPolyhedron<32> >(const hpmc_args_t& args,
                                                            const typename ShapeConvexPolyhedron<32> ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeConvexPolyhedron<64> >(const hpmc_args_t& args,
                                                            const typename ShapeConvexPolyhedron<64> ::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeConvexPolyhedron<128> >(const hpmc_args_t& args,
                                                            const typename ShapeConvexPolyhedron<128> ::param_type *d_params);

//! HPMC update for ShapeSpheropolyhedron
template hipError_t gpu_hpmc_update<ShapeSpheropolyhedron<8> >(const hpmc_args_t& args,
                                                            const typename ShapeSpheropolyhedron<8>::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeSpheropolyhedron<16> >(const hpmc_args_t& args,
                                                            const typename ShapeSpheropolyhedron<16>::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeSpheropolyhedron<32> >(const hpmc_args_t& args,
                                                            const typename ShapeSpheropolyhedron<32>::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeSpheropolyhedron<64> >(const hpmc_args_t& args,
                                                            const typename ShapeSpheropolyhedron<64>::param_type *d_params);
template hipError_t gpu_hpmc_update<ShapeSpheropolyhedron<128> >(const hpmc_args_t& args,
                                                            const typename ShapeSpheropolyhedron<128>::param_type *d_params);

//! HPMC update for ShapeSimplePolygon
template hipError_t gpu_hpmc_update<ShapeSimplePolygon>(const hpmc_args_t& args,
                                                         const typename ShapeSimplePolygon::param_type *d_params);

//! HPMC update for ShapeEllipsoid
template hipError_t gpu_hpmc_update<ShapeEllipsoid>(const hpmc_args_t& args,
                                                     const typename ShapeEllipsoid::param_type *d_params);

//! HPMC update for ShapeSpheropolygon
template hipError_t gpu_hpmc_update<ShapeSpheropolygon>(const hpmc_args_t& args,
                                                         const typename ShapeSpheropolygon::param_type *d_params);

//! HPMC update for ShapeFacetedSphere
template hipError_t gpu_hpmc_update<ShapeFacetedSphere>(const hpmc_args_t& args,
                                                        const typename ShapeFacetedSphere::param_type *d_params);

#ifdef ENABLE_SPHINX_GPU
//! HPMC update for ShapeSphinx
template hipError_t gpu_hpmc_update<ShapeSphinx>(const hpmc_args_t& args,
                                                  const typename ShapeSphinx::param_type *d_params);
#endif

//! HPMC update for ShapeUnion<ShapeSphere>
template hipError_t gpu_hpmc_update< ShapeUnion<ShapeSphere> >(const hpmc_args_t& args,
                                                  const typename ShapeUnion<ShapeSphere>::param_type *d_params);

}; // end namespace detail

} // end namespace hpmc
