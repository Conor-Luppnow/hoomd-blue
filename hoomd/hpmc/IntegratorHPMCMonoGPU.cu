#include "hip/hip_runtime.h"
// Copyright (c) 2009-2019 The Regents of the University of Michigan
// This file is part of the HOOMD-blue project, released under the BSD 3-Clause License.

#include "IntegratorHPMCMonoGPU.cuh"
#include "hoomd/RandomNumbers.h"

#include "hoomd/GPUPartition.cuh"
#include "hoomd/RandomNumbers.h"
#include "hoomd/RNGIdentifiers.h"
#include "hoomd/CachedAllocator.h"

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>


namespace hpmc
{
namespace gpu
{
namespace kernel
{

//! Kernel to generate expanded cells
/*! \param d_excell_idx Output array to list the particle indices in the expanded cells
    \param d_excell_size Output array to list the number of particles in each expanded cell
    \param excli Indexer for the expanded cells
    \param d_cell_idx Particle indices in the normal cells
    \param d_cell_size Number of particles in each cell
    \param d_cell_adj Cell adjacency list
    \param ci Cell indexer
    \param cli Cell list indexer
    \param cadji Cell adjacency indexer
    \param ngpu Number of active devices

    gpu_hpmc_excell_kernel executes one thread per cell. It gathers the particle indices from all neighboring cells
    into the output expanded cell.
*/
__global__ void hpmc_excell(unsigned int *d_excell_idx,
                            unsigned int *d_excell_size,
                            const Index2D excli,
                            const unsigned int *d_cell_idx,
                            const unsigned int *d_cell_size,
                            const unsigned int *d_cell_adj,
                            const Index3D ci,
                            const Index2D cli,
                            const Index2D cadji,
                            const unsigned int ngpu)
    {
    // compute the output cell
    unsigned int my_cell = 0;
    my_cell = blockDim.x * blockIdx.x + threadIdx.x;

    if (my_cell >= ci.getNumElements())
        return;

    unsigned int my_cell_size = 0;

    // loop over neighboring cells and build up the expanded cell list
    for (unsigned int offset = 0; offset < cadji.getW(); offset++)
        {
        unsigned int neigh_cell = d_cell_adj[cadji(offset, my_cell)];

        // iterate over per-device cell lists
        for (unsigned int igpu = 0; igpu < ngpu; ++igpu)
            {
            unsigned int neigh_cell_size = d_cell_size[neigh_cell+igpu*ci.getNumElements()];

            for (unsigned int k = 0; k < neigh_cell_size; k++)
                {
                // read in the index of the new particle to add to our cell
                unsigned int new_idx = d_cell_idx[cli(k, neigh_cell)+igpu*cli.getNumElements()];
                d_excell_idx[excli(my_cell_size, my_cell)] = new_idx;
                my_cell_size++;
                }
            }
        }

    // write out the final size
    d_excell_size[my_cell] = my_cell_size;
    }

//! Kernel for grid shift
/*! \param d_postype postype of each particle
    \param d_image Image flags for each particle
    \param N number of particles
    \param box Simulation box
    \param shift Vector by which to translate the particles

    Shift all the particles by a given vector.

    \ingroup hpmc_kernels
*/
__global__ void hpmc_shift(Scalar4 *d_postype,
                          int3 *d_image,
                          const unsigned int N,
                          const BoxDim box,
                          const Scalar3 shift)
    {
    // identify the active cell that this thread handles
    unsigned int my_pidx = blockIdx.x * blockDim.x + threadIdx.x;

    // this thread is inactive if it indexes past the end of the particle list
    if (my_pidx >= N)
        return;

    // pull in the current position
    Scalar4 postype = d_postype[my_pidx];

    // shift the position
    Scalar3 pos = make_scalar3(postype.x, postype.y, postype.z);
    pos += shift;

    // wrap the particle back into the box
    int3 image = d_image[my_pidx];
    box.wrap(pos, image);

    // write out the new position and orientation
    d_postype[my_pidx] = make_scalar4(pos.x, pos.y, pos.z, postype.w);
    d_image[my_pidx] = image;
    }

//!< Kernel to evaluate convergence
__global__ void hpmc_check_convergence(
                 const unsigned int *d_trial_move_type,
                 const unsigned int *d_reject_out_of_cell,
                 unsigned int *d_reject_in,
                 unsigned int *d_reject_out,
                 unsigned int *d_condition,
                 const unsigned int nwork,
                 const unsigned work_offset)
    {
    // the particle we are handling
    unsigned int work_idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (work_idx >= nwork)
        return;
    unsigned int i = work_idx + work_offset;

    // is this particle considered?
    bool move_active = d_trial_move_type[i] > 0;

    // combine with reject flag from gen_moves for particles which are always rejected
    bool reject = d_reject_out_of_cell[i] || d_reject_out[i];

    // did the answer change since the last iteration?
    if (move_active && reject != d_reject_in[i])
        {
        // flag that we're not done yet (a trivial race condition upon write)
        *d_condition = 1;
        }

    // update the reject flags
    d_reject_out[i] = reject;

    // clear input
    d_reject_in[i] = 0;
    }

//! Generate number of depletants per particle
__global__ void generate_num_depletants(const unsigned int seed,
                                        const unsigned int timestep,
                                        const unsigned int select,
                                        const unsigned int num_types,
                                        const unsigned int depletant_type_a,
                                        const unsigned int depletant_type_b,
                                        const Index2D depletant_idx,
                                        const unsigned int work_offset,
                                        const unsigned int nwork,
                                        const Scalar *d_lambda,
                                        const Scalar4 *d_postype,
                                        unsigned int *d_n_depletants)
    {
    unsigned int idx = threadIdx.x + blockDim.x*blockIdx.x;

    if (idx >= nwork)
        return;

    idx += work_offset;

    hoomd::RandomGenerator rng_poisson(hoomd::RNGIdentifier::HPMCDepletantNum, idx, seed, timestep,
        select*depletant_idx.getNumElements() + depletant_idx(depletant_type_a,depletant_type_b));
    Index2D typpair_idx(num_types);
    unsigned int type_i = __scalar_as_int(d_postype[idx].w);
    d_n_depletants[idx] = hoomd::PoissonDistribution<Scalar>(
        d_lambda[type_i*depletant_idx.getNumElements()+depletant_idx(depletant_type_a,depletant_type_b)])(rng_poisson);
    }

__global__ void hpmc_reduce_counters(const unsigned int ngpu,
                     const unsigned int pitch,
                     const hpmc_counters_t *d_per_device_counters,
                     hpmc_counters_t *d_counters,
                     const unsigned int implicit_pitch,
                     const Index2D depletant_idx,
                     const hpmc_implicit_counters_t *d_per_device_implicit_counters,
                     hpmc_implicit_counters_t *d_implicit_counters)
    {
    for (unsigned int igpu = 0; igpu < ngpu; ++igpu)
        {
        *d_counters = *d_counters + d_per_device_counters[igpu*pitch];

        for (unsigned int itype = 0; itype < depletant_idx.getNumElements(); ++itype)
            d_implicit_counters[itype] = d_implicit_counters[itype] + d_per_device_implicit_counters[itype+igpu*implicit_pitch];
        }
    }

} // end namespace kernel

//! Driver for kernel::hpmc_excell()
void hpmc_excell(unsigned int *d_excell_idx,
                 unsigned int *d_excell_size,
                 const Index2D& excli,
                 const unsigned int *d_cell_idx,
                 const unsigned int *d_cell_size,
                 const unsigned int *d_cell_adj,
                 const Index3D& ci,
                 const Index2D& cli,
                 const Index2D& cadji,
                 const unsigned int ngpu,
                 const unsigned int block_size)
    {
    assert(d_excell_idx);
    assert(d_excell_size);
    assert(d_cell_idx);
    assert(d_cell_size);
    assert(d_cell_adj);

    // determine the maximum block size and clamp the input block size down
    static int max_block_size = -1;
    if (max_block_size == -1)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::hpmc_excell)));
        max_block_size = attr.maxThreadsPerBlock;
        }

    // setup the grid to run the kernel
    dim3 threads(min(block_size, (unsigned int)max_block_size), 1, 1);
    dim3 grid(ci.getNumElements() / block_size + 1, 1, 1);

    hipLaunchKernelGGL(kernel::hpmc_excell, dim3(grid), dim3(threads), 0, 0, d_excell_idx,
                                           d_excell_size,
                                           excli,
                                           d_cell_idx,
                                           d_cell_size,
                                           d_cell_adj,
                                           ci,
                                           cli,
                                           cadji,
                                           ngpu);

    }

//! Kernel driver for kernel::hpmc_shift()
void hpmc_shift(Scalar4 *d_postype,
                int3 *d_image,
                const unsigned int N,
                const BoxDim& box,
                const Scalar3 shift,
                const unsigned int block_size)
    {
    assert(d_postype);
    assert(d_image);

    // setup the grid to run the kernel
    dim3 threads_shift(block_size, 1, 1);
    dim3 grid_shift(N / block_size + 1, 1, 1);

    hipLaunchKernelGGL(kernel::hpmc_shift, dim3(grid_shift), dim3(threads_shift), 0, 0, d_postype,
                                                      d_image,
                                                      N,
                                                      box,
                                                      shift);

    // after this kernel we return control of cuda managed memory to the host
    hipDeviceSynchronize();
    }


void hpmc_check_convergence(const unsigned int *d_trial_move_type,
                 const unsigned int *d_reject_out_of_cell,
                 unsigned int *d_reject_in,
                 unsigned int *d_reject_out,
                 unsigned int *d_condition,
                 const GPUPartition& gpu_partition,
                 const unsigned int block_size)
    {
    // determine the maximum block size and clamp the input block size down
    static int max_block_size = -1;
    if (max_block_size == -1)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::hpmc_check_convergence)));
        max_block_size = attr.maxThreadsPerBlock;
        }

    // setup the grid to run the kernel
    unsigned int run_block_size = min(block_size, (unsigned int)max_block_size);

    dim3 threads(run_block_size, 1, 1);

    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        unsigned int nwork = range.second - range.first;
        const unsigned int num_blocks = nwork/run_block_size + 1;
        dim3 grid(num_blocks, 1, 1);

        hipLaunchKernelGGL(kernel::hpmc_check_convergence, grid, threads, 0, 0,
            d_trial_move_type,
            d_reject_out_of_cell,
            d_reject_in,
            d_reject_out,
            d_condition,
            nwork,
            range.first);
        }
    }

void generate_num_depletants(const unsigned int seed,
                             const unsigned int timestep,
                             const unsigned int select,
                             const unsigned int num_types,
                             const unsigned int depletant_type_a,
                             const unsigned int depletant_type_b,
                             const Index2D depletant_idx,
                             const Scalar *d_lambda,
                             const Scalar4 *d_postype,
                             unsigned int *d_n_depletants,
                             const unsigned int block_size,
                             const hipStream_t *streams,
                             const GPUPartition& gpu_partition)
    {
    // determine the maximum block size and clamp the input block size down
    static unsigned int max_block_size = UINT_MAX;
    if (max_block_size == UINT_MAX)
        {
        hipFuncAttributes attr;
        hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(reinterpret_cast<const void*>(kernel::generate_num_depletants)));
        max_block_size = attr.maxThreadsPerBlock;
        }

    unsigned int run_block_size = min(block_size, max_block_size);

    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);
        unsigned int nwork = range.second - range.first;

        hipLaunchKernelGGL(kernel::generate_num_depletants, nwork/run_block_size+1, run_block_size, 0, streams[idev],
            seed,
            timestep,
            select,
            num_types,
            depletant_type_a,
            depletant_type_b,
            depletant_idx,
            range.first,
            nwork,
            d_lambda,
            d_postype,
            d_n_depletants);
        }
    }

void get_max_num_depletants(unsigned int *d_n_depletants,
                            unsigned int *max_n_depletants,
                            const hipStream_t *streams,
                            const GPUPartition& gpu_partition,
                            CachedAllocator& alloc)
    {
    assert(d_n_depletants);
    thrust::device_ptr<unsigned int> n_depletants(d_n_depletants);
    for (int idev = gpu_partition.getNumActiveGPUs() - 1; idev >= 0; --idev)
        {
        auto range = gpu_partition.getRangeAndSetGPU(idev);

        #ifdef __HIP_PLATFORM_HCC__
        max_n_depletants[idev] = thrust::reduce(thrust::hip::par(alloc).on(streams[idev]),
        #else
        max_n_depletants[idev] = thrust::reduce(thrust::cuda::par(alloc).on(streams[idev]),
        #endif
            n_depletants + range.first,
            n_depletants + range.second,
            0,
            thrust::maximum<unsigned int>());
        }
    }

void reduce_counters(const unsigned int ngpu,
                     const unsigned int pitch,
                     const hpmc_counters_t *d_per_device_counters,
                     hpmc_counters_t *d_counters,
                     const unsigned int implicit_pitch,
                     const Index2D depletant_idx,
                     const hpmc_implicit_counters_t *d_per_device_implicit_counters,
                     hpmc_implicit_counters_t *d_implicit_counters)
    {
    hipLaunchKernelGGL(kernel::hpmc_reduce_counters, 1, 1, 0, 0,
                     ngpu,
                     pitch,
                     d_per_device_counters,
                     d_counters,
                     implicit_pitch,
                     depletant_idx,
                     d_per_device_implicit_counters,
                     d_implicit_counters);
    }

} // end namespace gpu
} // end namespace hpmc

